#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"



static void CatchError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define CATCH_ERROR( err ) (CatchError( err, __FILE__, __LINE__ ))


__global__ void kernel(double *res, double *d_arr, long long int n)
{
    long long int tid = blockIdx.x * blockDim.x + threadIdx.x;
    long long int offset = blockDim.x * gridDim.x;
    
    while (tid < n) {
        // write result to array
        res[tid] = tanf( d_arr[tid%90] * 3.141592653589 / 180 );
        tid += offset;
    }
}

void my_tan(double *res, double *d_arr, long long int n) {
    long long int tid = 0;

    while (tid < n) {
        // write result to array
        res[tid] = tanf(d_arr[tid % 90] * 3.141592653589 / 180);
        tid += 1;
    }
}

int main()
{
    long long int n = 1000000;

    hipEvent_t time_of_start, time_of_end;
    float res_timer_gpu;
    CATCH_ERROR( hipEventCreate( &time_of_start ) );
    CATCH_ERROR( hipEventCreate( &time_of_end ) );
    


    double d_arr[90];    // array with degrees
    for (int i = 0; i < 90; i++) {  // 0 to 90
        d_arr[i] = i;
    }

    double *res = (double*)malloc(n * sizeof(double));
    double *ar_d_dev, *res_dev;
    
    CATCH_ERROR( hipMalloc( &res_dev, n * sizeof(double) ) );
    CATCH_ERROR( hipMalloc( &ar_d_dev, 90 * sizeof(double) ) );

    CATCH_ERROR( hipMemcpy( ar_d_dev, d_arr, 90 * sizeof(double), hipMemcpyHostToDevice ) );
    

    CATCH_ERROR( hipEventRecord( time_of_start ) );

    kernel <<<256,256>>>(res_dev, ar_d_dev, n);

    CATCH_ERROR( hipEventRecord( time_of_end ));
    CATCH_ERROR( hipEventSynchronize( time_of_end ) );
    
    CATCH_ERROR( hipEventElapsedTime( &res_timer_gpu, time_of_start, time_of_end ) );
    
    CATCH_ERROR( hipEventDestroy( time_of_start ) );
    CATCH_ERROR( hipEventDestroy( time_of_end ) );

    CATCH_ERROR( hipMemcpy( res, res_dev, n * sizeof(double), hipMemcpyDeviceToHost ) );
    
    CATCH_ERROR( hipFree( ar_d_dev ) );
    CATCH_ERROR( hipFree( res_dev ) );

    for (long long i = 0; i < n; i++) {
        printf("%f\n", res[i]);
    }

    free(res);

    res = (double*)malloc(n * sizeof(double));
    double res_timer_cpu = 0.0;

    clock_t begin = clock();
    
    my_tan(res, d_arr, n);

    clock_t end = clock();

    res_timer_cpu += (double)(end - begin) / CLOCKS_PER_SEC;

    printf("CPU time: %f\n", res_timer_cpu*1000);

    printf("GPU time: %f\n", res_timer_gpu);
    
    free(res);
    return 0;
}
